#include "hip/hip_runtime.h"
/*
 * MO644 Projeto Final 
 * 
 * Renan Gomes Pereira 103927
 * 
 * Conway's Game of Life
 * 
 * CUDA shared memory version
 */

extern "C" {
    #include "conway_functions.h"
}

#define TILE_WIDTH 16

// for shared memory
#define MASK_WIDTH 3
#define RADIUS (MASK_WIDTH-1)/2
#define SM_LINE_SIZE (TILE_WIDTH+MASK_WIDTH-1)
#define SM_SIZE SM_LINE_SIZE*SM_LINE_SIZE

extern char *board;
extern char *temp;
extern int nrows, ncols;

// cuda variables
char *d_board;
char *d_temp;
int board_size;

// allocates and initialize cuda board and variables
void initialize_cuda_board() {
    board_size = sizeof(char)*nrows*ncols;
    
    hipMalloc((void **) &d_board, board_size);
    hipMalloc((void **) &d_temp, board_size);
    
    hipMemcpy(d_board, board, board_size, hipMemcpyHostToDevice);
    hipMemcpy(d_temp, temp, board_size, hipMemcpyHostToDevice);
}

__device__ inline int num_neighbours_cuda(char *board) {
    int num_adj = 0;
    int i,j;
    
    for(i = threadIdx.y; i < MASK_WIDTH+threadIdx.y; i++) {
        for(j = threadIdx.x; j < MASK_WIDTH+threadIdx.x; j++) {
            if(board[i*SM_LINE_SIZE + j] == ON)
                num_adj++; 
        }
    }
        
    return num_adj;
}

__global__ void copy_temp_to_board(char *board, char *temp, int nrows, int ncols) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int id = col + row*ncols;
    
    if (row < nrows && col < ncols) {
        board[id] = temp[id];
    }
}

__global__ void update_board_cuda(char *board, char *temp, int nrows, int ncols) {
    __shared__ char shared_board_part[SM_SIZE];
    
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int id = col + row*ncols;
    int neighbours, curr_r, curr_c, mapID;
    
    // fills the shared memory array
	for(curr_r=row-RADIUS; curr_r-row+RADIUS+threadIdx.y < MASK_WIDTH-1+TILE_WIDTH; curr_r += TILE_WIDTH) {	
		for(curr_c=col-RADIUS; curr_c-col+RADIUS+threadIdx.x < MASK_WIDTH-1+TILE_WIDTH; curr_c += TILE_WIDTH) {
			mapID = SM_LINE_SIZE*(curr_r-row+RADIUS + threadIdx.y) + curr_c-col+RADIUS + threadIdx.x;
			
			if(curr_c >= 0 && curr_c < ncols && curr_r >= 0 && curr_r < nrows) {
                shared_board_part[mapID] = board[curr_r*ncols + curr_c];
			}
			
			// borders
			else {
				shared_board_part[mapID] = 0;
			}
		}
	}

	__syncthreads();
    
    if (row < nrows && col < ncols) {
        neighbours = num_neighbours_cuda(shared_board_part);
        
        // a cell is not a neighbour of itself 
        if(board[id] == ON) neighbours--;
        
        /* Dies by underpopulation. */
        if (neighbours < 2 && board[id] == ON) {
            temp[id] = OFF; 
        } 
        /* Dies by overpopulation. */
        else if (neighbours > 3 && board[id] == ON) {
            temp[id] = OFF; 
        }
        
        /* Become alive because of reproduction. */
        else if (neighbours == 3 && board[id] == OFF) {
            temp[id] = ON;
        }
        
        /* Otherwise the cell lives with just the right company. */
        else {
            temp[id] = board[id];
        }
	}
}

void update_board(int n, int nt) {
    printf("Running CUDA shared!\n");
    
    // switch boards so we dont have to copy temp to board every time
    int switch_boards = 0;
    
    initialize_cuda_board();
    
    dim3 dimGrid(ceil(ncols/(float)TILE_WIDTH), ceil(nrows/(float) TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    
    for(int it = 0; it < n; it++) {
        if(switch_boards) {
            update_board_cuda<<<dimGrid,dimBlock>>>(d_board, d_temp, nrows, ncols);
            switch_boards = 0;
        }
        
        else {
            update_board_cuda<<<dimGrid,dimBlock>>>(d_temp, d_board, nrows, ncols);
            switch_boards = 1;
        }
        
        hipDeviceSynchronize();
    }
    
    // copies the result back to the host
    if(n%2 != 0) {
        hipMemcpy(board, d_board, board_size, hipMemcpyDeviceToHost);
    }
    
    else {
        hipMemcpy(board, d_temp, board_size, hipMemcpyDeviceToHost);
        copy_temp_to_board<<<dimGrid,dimBlock>>>(d_board, d_temp, nrows, ncols);
    }
    
    hipFree(d_board);
    hipFree(d_temp);
}
