#include "hip/hip_runtime.h"
/*
 * MO644 Projeto Final 
 * 
 * Renan Gomes Pereira 103927
 * Pedro Augusto Gebin de Carvalho 208618 
 * 
 * Conway's Game of Life
 * 
 * CUDA version
 */

extern "C" {
    #include "conway_functions.h"
}
#include <stdio.h>
#include <string.h>

// TODO TESTAR VARIOS TILE WIDTH PARA O RELATORIO
#define TILE_WIDTH 16

extern char *board;
extern char *temp;
extern int nrows, ncols;

// cuda variables
char *d_board;
char *d_temp;
int board_size;

// update the board according to the game of life rules
void update_board();

// allocates and initialize cuda board and variables
void initialize_cuda_board();

int main(void) {
    double t_start, t_end;
    
    int n; // number of iteractions
    int nt; // number of threads (not used in cuda version)
    
    // read input
    scanf("%d %d", &n, &nt);
    scanf("%d %d",&nrows, &ncols);
    
    initialize_board();

    #ifdef COMPARE_SERIAL
        int n2 = n;
        initialize_board_2();
    #endif
    
    // run n iterations
    t_start = rtclock();
    initialize_cuda_board();
    while(n--) update_board();
    // copies the result back to the host
    hipMemcpy(board, d_board, board_size, hipMemcpyDeviceToHost);
    hipFree(d_board);
    hipFree(d_temp);
    t_end = rtclock();
    
    double t_time = t_end - t_start;
   
    #ifdef PRINT_BOARD
        print_board();
    #endif
    
    printf("Time: %f seconds\n", t_time);
    
    // Run serial version and compare with parallel results
    // Prints the speedup
    #ifdef COMPARE_SERIAL
        copy_board2_to_temp();
        
        t_start = rtclock();
        while(n2--) update_board_serial();
        t_end = rtclock();
        double t_time_serial = t_end - t_start;
        printf("Time serial: %f seconds\n", t_time_serial);
        printf("Speedup: %f seconds\n", t_time_serial/t_time);
        
        int diff = compare_serial_parallel();
        if(diff == 0) printf("Same result!\n");
        else printf("ERROR: Different result! Number of differences = %d\n", diff);
    #endif

    free_board();
   
    return 0;
}

__device__ int num_neighbours_cuda(char * board, int row, int col, int nrows, int ncols) {
    int num_adj = 0;
    int i,j;
    
    for(i=row-1;i<=row+1;i++) {
        for(j=col-1;j<=col+1;j++) {
            if(i==row && j == col) continue;
            
            if(i >= 0 && j>=0 && i < nrows && j < ncols && board[i*ncols+j] == ON)
                num_adj++;  
        }
    }
    
    return num_adj;
}

__global__ void update_board_cuda(char *board, char *temp, int nrows, int ncols) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int id = col + row*ncols;
    int neighbours;
    
    if (row < nrows && col < ncols) {
        neighbours = num_neighbours_cuda(board, row, col, nrows, ncols);
            
        /* Dies by underpopulation. */
        if (neighbours < 2 && board[id] == ON) {
            temp[id] = OFF; 
        } 
        /* Dies by overpopulation. */
        else if (neighbours > 3 && board[id] == ON) {
            temp[id] = OFF; 
        }
        
        /* Become alive because of reproduction. */
        else if (neighbours == 3 && board[id] == OFF) {
            temp[id] = ON;
        }
        
        /* Otherwise the cell lives with just the right company. */
	}
}

void initialize_cuda_board() {
    board_size = sizeof(char)*nrows*ncols;
    
    hipMalloc((void **) &d_board, board_size);
    hipMalloc((void **) &d_temp, board_size);
    
    hipMemcpy(d_board, board, board_size, hipMemcpyHostToDevice);
    hipMemcpy(d_temp, temp, board_size, hipMemcpyHostToDevice);
}

void update_board() {
    dim3 dimGrid(ceil(ncols/(float)TILE_WIDTH), ceil(nrows/(float) TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    
    update_board_cuda<<<dimGrid,dimBlock>>>(d_board, d_temp, ncols, nrows);
    // TODO TESTAR SEM
    hipDeviceSynchronize();
}
