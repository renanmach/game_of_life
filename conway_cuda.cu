#include "hip/hip_runtime.h"
/*
 * MO644 Projeto Final 
 * 
 * Renan Gomes Pereira 103927
 * Pedro Augusto Gebin de Carvalho 208618 
 * 
 * Conway's Game of Life
 * 
 * CUDA version
 */

extern "C" {
    #include "conway_functions.h"
}

// TODO TESTAR VARIOS TILE WIDTH PARA O RELATORIO ***********************************
#define TILE_WIDTH 16

extern char *board;
extern char *temp;
extern int nrows, ncols;

// cuda variables
char *d_board;
char *d_temp;
int board_size;

// allocates and initialize cuda board and variables
void initialize_cuda_board() {
    board_size = sizeof(char)*nrows*ncols;
    
    hipMalloc((void **) &d_board, board_size);
    hipMalloc((void **) &d_temp, board_size);
    
    hipMemcpy(d_board, board, board_size, hipMemcpyHostToDevice);
    hipMemcpy(d_temp, temp, board_size, hipMemcpyHostToDevice);
}

__device__ int num_neighbours_cuda(char *board, int row, int col, int nrows, int ncols) {
    int num_adj = 0;
    int i,j;
    
    for(i=row-1;i<=row+1;i++) {
        for(j=col-1;j<=col+1;j++) {
            // check boundaries and if the neighbour is alive
            if(i >= 0 && j>=0 && i < nrows && j < ncols && board[i*ncols+j] == ON)
                num_adj++;  
        }
    }
    
    // a cell is not a neighbour of itself 
    if(board[row*ncols+col] == ON)
        num_adj--; 
    
    return num_adj;
}

__global__ void copy_temp_to_board(char *board, char *temp, int nrows, int ncols) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int id = col + row*ncols;
    
    if (row < nrows && col < ncols) {
        board[id] = temp[id];
    }
}

__global__ void update_board_cuda(char *board, char *temp, int nrows, int ncols) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int id = col + row*ncols;
    int neighbours;
    
    if (row < nrows && col < ncols) {
        neighbours = num_neighbours_cuda(board, row, col, nrows, ncols);
            
        /* Dies by underpopulation. */
        if (neighbours < 2 && board[id] == ON) {
            temp[id] = OFF; 
        } 
        /* Dies by overpopulation. */
        else if (neighbours > 3 && board[id] == ON) {
            temp[id] = OFF; 
        }
        
        /* Become alive because of reproduction. */
        else if (neighbours == 3 && board[id] == OFF) {
            temp[id] = ON;
        }
        
        /* Otherwise the cell lives with just the right company. */
        else {
            temp[id] = board[id];
        }
	}
}

void update_board(int n, int nt) {
    printf("Running CUDA!\n");
    
    // switch boards so we dont have to copy temp to board every time
    int switch_boards = 0;
    
    initialize_cuda_board();
    
    dim3 dimGrid(ceil(ncols/(float)TILE_WIDTH), ceil(nrows/(float) TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    
    for(int it = 0; it < n; it++) {
        if(switch_boards) {
            update_board_cuda<<<dimGrid,dimBlock>>>(d_board, d_temp, nrows, ncols);
            switch_boards = 0;
        }
        
        else {
            update_board_cuda<<<dimGrid,dimBlock>>>(d_temp, d_board, nrows, ncols);
            switch_boards = 1;
        }
        
        hipDeviceSynchronize();
        
        //copy_temp_to_board<<<dimGrid,dimBlock>>>(d_board, d_temp, nrows, ncols);
    }
    
    
    
    // copies the result back to the host
    if(n%2 != 0) {
        hipMemcpy(board, d_board, board_size, hipMemcpyDeviceToHost);
    }
    
    else {
        hipMemcpy(board, d_temp, board_size, hipMemcpyDeviceToHost);
        copy_temp_to_board<<<dimGrid,dimBlock>>>(d_board, d_temp, nrows, ncols);
    }
    
    hipFree(d_board);
    hipFree(d_temp);
}
